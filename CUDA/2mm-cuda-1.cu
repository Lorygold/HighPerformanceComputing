#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include CUDA */
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "2mm.h"
int N = NI;

/* Array initialization. */
static void init_array(int ni, int nj, int nk, int nl,
                       DATA_TYPE *alpha,  
                       DATA_TYPE *beta,
                       DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nl),
                       DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
                       DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
                       DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
{
  int i, j;

  *alpha = 32412;
  *beta = 2123;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++)
      A[i][j] = ((DATA_TYPE)i * j) / ni;
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i][j] = ((DATA_TYPE)i * (j + 1)) / nj;
  for (i = 0; i < nl; i++)
    for (j = 0; j < nj; j++)
      C[i][j] = ((DATA_TYPE)i * (j + 3)) / nl;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++)
      D[i][j] = ((DATA_TYPE)i * (j + 2)) / nk;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nl,
                        DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, D[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

__global__ void kernelCUDA1(DATA_TYPE * __restrict__ tmp, DATA_TYPE * __restrict__ A, DATA_TYPE * __restrict__ B, DATA_TYPE alfa, DATA_TYPE _PB_NI, DATA_TYPE _PB_NJ, DATA_TYPE _PB_NK){
  // for (i = 0; i < _PB_NI; i++){
  //     for (j = 0; j < _PB_NJ; j++)
  //     {
  //       tmp[i][j] = 0;
  //       for (k = 0; k < _PB_NK; ++k)
  //         tmp[i][j] += alpha * A[i][k] * B[k][j];
  //     }
  // }
  // Get row and column indices
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Check bounds
  if (i < ni && j < nj) {
    // Initialize result
    DATA_TYPE sum = 0;
    // Loop over k
    for (int k = 0; k < nk; k++) {
      // Accumulate product
      sum += alpha * A[i * nk + k] * B[k * nj + j];
    }
    // Write result
    tmp[i * nj + j] = sum;
  }

}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_2mm(int ni, int nj, int nk, int nl,
                       DATA_TYPE alpha,
                       DATA_TYPE beta,
                       DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
                       DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
                       DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
                       DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
                       DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
{
  int i, j, k;
  // Allocate device memory
  hipMalloc((void **)&d_a, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc((void **)&d_b, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc((void **)&d_c, sizeof(DATA_TYPE) * NL * NJ);
  hipMalloc((void **)&d_d, sizeof(DATA_TYPE) * NI * NL);
  hipMalloc((void **)&d_tmp, sizeof(DATA_TYPE) * NI * NJ);
  // Data copy from host to device
  hipMemcpy(d_a, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy(d_c, C, sizeof(DATA_TYPE) * NL * NJ, hipMemcpyHostToDevice);
  hipMemcpy(d_d, D, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);
  dim3 grid()
  dim3 block((NI))
  /* D := alpha*A*B*C + beta*D */
  kernelCUDA1<<<grid,block>>>(tmp, A, B, alfa, _PB_NI, _PB_NJ, _PB_NK);
  hipMemcpy(tmp, d_tmp, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);
  printf("ciao");
    // for (i = 0; i < _PB_NI; i++)
    //   for (j = 0; j < _PB_NL; j++)
    //   {
    //     D[i][j] *= beta;
    //     for (k = 0; k < _PB_NJ; ++k)
    //       D[i][j] += tmp[i][k] * C[k][j];
    //   }
}

int main(int argc, char **argv)
{
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;
  int nl = NL;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(tmp, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
  POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NL, NJ, nl, nj);
  POLYBENCH_2D_ARRAY_DECL(D, DATA_TYPE, NI, NL, ni, nl);

  /* Initialize array(s). */
  init_array(ni, nj, nk, nl, &alpha, &beta,
             POLYBENCH_ARRAY(A),
             POLYBENCH_ARRAY(B),
             POLYBENCH_ARRAY(C),
             POLYBENCH_ARRAY(D));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_2mm(ni, nj, nk, nl,
             alpha, beta,
             POLYBENCH_ARRAY(tmp),
             POLYBENCH_ARRAY(A),
             POLYBENCH_ARRAY(B),
             POLYBENCH_ARRAY(C),
             POLYBENCH_ARRAY(D));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(ni, nl, POLYBENCH_ARRAY(D)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(tmp);
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(D);

  return 0;
}
