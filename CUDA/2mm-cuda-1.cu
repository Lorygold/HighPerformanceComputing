#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include CUDA */
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "2mm.h"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE (32)
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/* Array initialization. */
static void init_array(int ni, int nj, int nk, int nl,
                       double *alpha,  
                       double *beta,
                       double* A,
                       double* B,
                       double* C,
                       double* D)
{
  int i, j;
  //printf("ni=%d nk=%d nj=%d nl=%d\n", ni, nk, nj, nl);
  *alpha = 32412;
  *beta = 2123;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++){
      A[i * nk + j] = (i * j) / (double)ni;
    }
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i * nj + j] = (i * (j + 1)) / (double)nj;
  for (i = 0; i < nl; i++)
    for (j = 0; j < nj; j++)
      C[i * nj + j] = (i * (j + 3)) / (double)nl;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++)
      D[i * nl + j] = (i * (j + 2)) / (double)nk;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nl,
                        DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, D[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

__global__ void kernelCUDA1(double* tmp, double* A, double* B, double alpha, int ni, int nj, int nk)
{
  // Get row and column indices
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Check bounds
  if (i < ni && j < nj) {
    // Initialize result
    double sum = 0.0;
    // Loop over k
    for (int k = 0; k < nk; k++){
      // Accumulate product
      sum += alpha * A[(i * nk) + k] * B[(k * nj) + j];
    }
    // Write result
    tmp[(i * nj) + j] = sum;
  }
}

__global__ void kernelCUDA2(double* tmp, double* D, double* C, double beta, int ni, int nj, int nl)
{

  // Get row and column indices
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  // Check bounds
  if (i < ni && j < nj) {
    
   double sum = 0.0; 
   for (int k = 0; k < nl; k++){
      // Accumulate product
      sum += tmp[(i * nl) + k] * C[(k * nl) + j];
    }
    // Write result
    D[(i * nl) + j] = sum*beta;
  }
}


int main(int argc, char **argv)
{
  
  int N = NI;
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;
  int nl = NL;
  /* Variable declaration/allocation. */
  double alpha;
  double beta;
  double* A = new double[ni*nk];
  double* B = new double[nk*nj];
  double* C = new double[nl*nj];
  double* D = new double[ni*nl];
  double* tmp = new double[ni*nj];

  /* Initialize array(s). */
  init_array(ni, nj, nk, nl, &alpha, &beta,A,B,C,D);

  /* Start timer. */
  polybench_start_instruments;

  // Allocate device memory
  double *d_a, *d_b, *d_c, *d_d, *d_tmp;
  gpuErrchk(hipMalloc((void **)&d_a, sizeof(double) * ni * nk));
  gpuErrchk(hipMalloc((void **)&d_b, sizeof(double) * nk * nj));
  gpuErrchk(hipMalloc((void **)&d_c, sizeof(double) * nl * nj));
  gpuErrchk(hipMalloc((void **)&d_d, sizeof(double) * ni * nl));
  gpuErrchk(hipMalloc((void **)&d_tmp, sizeof(double) * ni * nj));
  //printf("N=%d, BLOCK_SIZE=%d grid_size=%d\n",N,BLOCK_SIZE,((N+(BLOCK_SIZE-1))/BLOCK_SIZE));
  // Data copy from host to device
  gpuErrchk(hipMemcpy(d_a, A, sizeof(double) * ni * nk, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_b, B, sizeof(double) * nk * nj, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_c, C, sizeof(double) * nl * nj, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_d, D, sizeof(double) * ni * nl, hipMemcpyHostToDevice));

  dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid_size((N+BLOCK_SIZE-1) / (BLOCK_SIZE),(N+BLOCK_SIZE-1) / (BLOCK_SIZE));
  /* D := alpha*A*B*C + beta*D */
  printf("\ngrid_size=%d, block_size=%d\n",((N+BLOCK_SIZE-1) / (BLOCK_SIZE)) * ((N+BLOCK_SIZE-1) / (BLOCK_SIZE)), BLOCK_SIZE*BLOCK_SIZE );
  kernelCUDA1<<<grid_size,block_size>>>(d_tmp, d_a, d_b, alpha, ni, nj, nk);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  kernelCUDA2<<<grid_size,block_size>>>(d_tmp, d_d, d_c, beta, ni, nj, nl);
  gpuErrchk(hipMemcpy(tmp, d_tmp, sizeof(double) * ni * nj, hipMemcpyDeviceToHost));
  for(int z = 0; z < ni*nj; z++){
    printf("value=%f\n",tmp[z]);
  }
  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;


  /* Be clean. */
  delete[] tmp;
  delete[] A;
  delete[] B;
  delete[] C;
  delete[] D;
  hipFree(d_a);
  hipFree(d_b);
  //hipFree(d_c)
  //hipFree(d_d)
  hipFree(d_tmp);

  return 0;
}
